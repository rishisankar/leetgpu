#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

/*
Implementation of Flash Attention 2
https://arxiv.org/pdf/2307.08691

Q: Mxd
K: Nxd
V: Nxd
O: Mxd
l,m: Mx1
*/

constexpr int SRAM_SIZE = 4000;
// To store the vectors li and mi, we need at most
// d entries, which is at most 1024
constexpr int MAX_VECTOR_SIZE = 1024;
constexpr float NEGATIVE_INF = -1e20;

int ceildiv(int a, int b) {
    return (a + b - 1) / b;
}

/**
 * Load a block of the matrix from src to dst.
 * src intended to be global, dst intended to be shared memory.
 * Matrix is size MxN
 */
__device__ void matrix_block_load(
    float* dst, 
    const float* src, 
    int M,
    int N,
    int block_size,
    int block_idx
) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    int num_elts = M * N;
    int block_start = block_idx * block_size * N;
    int block_end = block_start + block_size * N;
    for (int i = block_start + tid; i < block_end; i += num_threads) {
        dst[i - block_start] = (i < num_elts) ? src[i] : 0;
    }
}

/**
 * Load a block of the matrix from src to dst, and transpose it.
 * src intended to be global, dst intended to be shared memory.
 * Matrix is size MxN
 */
 __device__ void matrix_block_load_transpose(
    float* dst, // will be size loop_block_size x N but transposed
    const float* src, 
    int M,
    int N,
    int block_size,
    int loop_block_size,
    int block_idx
) {

    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    int num_elts = M * N;
    int block_start = block_idx * block_size * N;
    int block_end = block_start + block_size * N;
    for (int i = block_start + tid; i < block_end; i += num_threads) {
        int r = (i - block_start) / N;
        int c = i % N;
        dst[c * loop_block_size + r] = (i < num_elts) ? src[i] : 0;
    }
}

/**
 * Store src into a block of dst.
 * src intended to be shared memory, dst intended to be global.
 * dst is size M x N, src is size block_size x N.
 */
__device__ void matrix_block_store(
    float* dst, 
    const float* src, 
    int M,
    int N,
    int block_size,
    int block_idx
) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    int block_start = block_idx * block_size * N;
    int block_end = min(M * N, block_start + block_size * N);
    for (int i = block_start + tid; i < block_end; i += num_threads) {
        dst[i] = src[i - block_start];
    }
}

/**
 * Fill array of size N with fill_value.
 */
__device__ void array_fill(
    float* array,
    float fill_value,
    int N
) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    for (int i = tid; i < N; i += num_threads) {
        array[i] = fill_value;
    }
}

/**
 * Computes matrix multiplication A*BT.
 * A is of size MxK, B is of size NxK.
 * Output C is of size MxN.
 * If add_to_output, A*BT is added to C instead of overwriting it.
 * This is a simple version, not optimized for speed.
 */
template <bool add_to_output = false>
__device__ void matrix_multiply(
    const float* A,
    const float* B,
    float* C, 
    int M,
    int N,
    int K
) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    int num_elts = M * N;
    for (int i = tid; i < num_elts; i += num_threads) {
        int m = i / N;
        int n = i % N;
        float sum = 0;
        for (int k = 0; k < K; ++k) {
            sum += A[m * K + k] * B[n * K + k];
        }
        if constexpr (add_to_output) {
            C[i] += sum;
        } else {
            C[i] = sum;
        }
    }
}

__device__ void divide_by_scalar(
    float* array,
    float scalar,
    int N
) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    for (int i = tid; i < N; i += num_threads) {
        array[i] /= scalar;
    }
}

/**
 * Assigns mi_cur to max(mi_prev, rowmax(Si)).
 * mi_cur / mi_prev are vectors of size Br in smem.
 * Si is a matrix of size Br x Bc in smem.
 */
__device__ void mi_update(
    float* mi_cur,
    const float* mi_prev,
    const float* Si,
    int Br,
    int Bc
) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    for (int i = tid; i < Br; i += num_threads) {
        float max_val = mi_prev[i];
        for (int j = 0; j < Bc; ++j) {
            max_val = max(max_val, Si[i * Bc + j]);
        }
        mi_cur[i] = max_val;
    }
}

/**
 * Converts Si to Pi, where Pi = exp(Si - mi).
 * Si is a matrix of size Br x Bc in smem.
 * Pi is a matrix of size Br x Bc in smem.
 * mi is a vector of size Br in smem.
 */
__device__ void si_to_pi(
    float* SiPi,
    const float* mi,
    int Br,
    int Bc
) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    for (int i = tid; i < Br * Bc; i += num_threads) {
        int r = i / Bc;
        SiPi[i] = exp(SiPi[i] - mi[r]);
    }
}

/**
 * Update li to exp(mi_prev - mi_cur) * li + rowsum(Pi).
 * li is a vector of size Br in smem.
 * Pi is a matrix of size Br x Bc in smem.
 * mi_prev is a vector of size Br in smem.
 * mi_cur is a vector of size Br in smem.
 */
__device__ void li_update(
    float* li,
    const float* Pi,
    const float* mi_prev,
    const float* mi_cur,
    int Br,
    int Bc
) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    for (int i = tid; i < Br; i += num_threads) {
        float sum = 0;
        for (int j = 0; j < Bc; ++j) {
            sum += Pi[i * Bc + j];
        }
        li[i] = exp(mi_prev[i] - mi_cur[i]) * li[i] + sum;
    }
}

/**
 * Update Oi to diag(exp(mi_prev - mi_cur)) * Oi + Pi * V.
 * Oi is a matrix of size Br x d in smem.
 * Pi is a matrix of size Br x Bc in smem.
 * V is a matrix of size Bc x d in smem.
 * mi_prev, mi_cur are vectors of size Br in smem.
 */
__device__ void Oi_update(
    float* Oi,
    const float* Pi,
    const float* VT,
    const float* mi_prev,
    const float* mi_cur,
    int Br,
    int Bc,
    int d
) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    int num_elts = Br * d;
    for (int i = tid; i < num_elts; i += num_threads) {
        int r = i / d;
        Oi[i] = exp(mi_prev[r] - mi_cur[r]) * Oi[i];
    }
    matrix_multiply<true>(Pi, VT, Oi, Br, d, Bc);
}

/**
 * Divide each row of Oi by that value of li.
 * Oi is a matrix of size Br x d in smem.
 * li is a vector of size Br in smem.
 */
__device__ void Oi_scale(
    float* Oi,
    const float* li,
    int Br,
    int d
) {
    int tid = threadIdx.x;
    int num_threads = blockDim.x;
    int num_elts = Br * d;
    for (int i = tid; i < num_elts; i += num_threads) {
        int r = i / d;
        Oi[i] /= li[r];
    }
}

__global__ void flash_attention_2_kernel(
    const float* Q, 
    const float* K, 
    const float* V, 
    float* O, 
    const int M, 
    const int N, 
    const int d, 
    const int Br, 
    const int Bc, 
    const int Tr, 
    const int Tc,
    const int alloc_size
) {
    extern __shared__ float s[];
    float *Oi = s;
    float *Qi = &s[alloc_size];
    // will first store Ki, then get overriden to ViT
    float *KiVi = &s[2 * alloc_size];
    // will first store Si, then get overriden to Pi
    float *SiPi = &s[3 * alloc_size];
    float *li = &s[4 * alloc_size];
    float *mi = &s[4 * alloc_size + MAX_VECTOR_SIZE];
    float *mi2 = &s[4 * alloc_size + 2 * MAX_VECTOR_SIZE];

    float* mi_prev = mi; // m(i,j-1)
    float* mi_cur = mi2; // m(i,j)

    for (int i = 0; i < Tr; i++) {
        int loopBr = min(Br, M - i * Br);
        matrix_block_load(Qi, Q, M, d, Br, i);
        array_fill(Oi, 0, loopBr * d);
        array_fill(li, 0, loopBr);
        array_fill(mi_prev, NEGATIVE_INF, loopBr);
        __syncthreads();
        for (int j = 0; j < Tc; j++) {
            int loopBc = min(Bc, N - j * Bc);
            matrix_block_load(KiVi, K, N, d, Bc, j);
            __syncthreads();
            matrix_multiply(Qi, KiVi, SiPi, loopBr, loopBc, d);
            __syncthreads();
            divide_by_scalar(SiPi, sqrtf(d), loopBr * loopBc);
            __syncthreads();
            mi_update(mi_cur, mi_prev, SiPi, loopBr, loopBc);
            __syncthreads();
            si_to_pi(SiPi, mi_cur, loopBr, loopBc);
            __syncthreads();
            li_update(li, SiPi, mi_prev, mi_cur, loopBr, loopBc);
            matrix_block_load_transpose(KiVi, V, N, d, Bc, loopBc, j);
            __syncthreads();
            Oi_update(Oi, SiPi, KiVi, mi_prev, mi_cur, loopBr, loopBc, d);
            __syncthreads();

            // swap mi_prev / mi_cur
            auto tmp = mi_prev;
            mi_prev = mi_cur;
            mi_cur = tmp;
        }
        Oi_scale(Oi, li, loopBr, d);
        __syncthreads();
        matrix_block_store(O, Oi, M, d, Br, i);
        __syncthreads();
    }
}

// Q, K, V, output are device pointers
void solve(const float* Q, const float* K, const float* V, float* output, int M, int N, int d) {
    int Bc = ceildiv(SRAM_SIZE, 4 * d);
    int Br = min(Bc, d);
    int Tr = ceildiv(M, Br);
    int Tc = ceildiv(N, Bc);

    int alloc_size = max(Br * Bc, Bc * d);
    int shmem_needed = (4 * alloc_size + 3 * MAX_VECTOR_SIZE) * sizeof(float);

    // call kernel
    const int threadsPerBlock = 1024;
    const int blocksPerGrid = 1;
    flash_attention_2_kernel<<<blocksPerGrid, threadsPerBlock, shmem_needed>>>(
        Q, K, V, output, M, N, d, Br, Bc, Tr, Tc, alloc_size
    );
}
