#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

constexpr int BLOCK_SIZE = 32;

__global__ void matrix_transpose_kernel(const float* input, float* output, int rows, int cols) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i < cols && j < rows) {
        output[i * rows + j] = input[j * cols + i];
    }
}


// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int rows, int cols) {
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 blocksPerGrid((cols + BLOCK_SIZE - 1) / BLOCK_SIZE,
                       (rows + BLOCK_SIZE - 1) / BLOCK_SIZE);

    matrix_transpose_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, rows, cols);
    hipDeviceSynchronize();
}
