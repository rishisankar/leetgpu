#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

using namespace nvcuda;

constexpr int BW = 64;
constexpr int NUM_WARPS = 16;

/*
Approach:
- uses tensor cores to multiply 16x16 matrices
- 16 warps per block each handling a 16x16 region
- blocks handle a 64x64 tile of output
*/

template <bool isKEven>
__global__ void matmul_kernel(const half* A, const half* B, half* C, int M, int N, int K, float alpha, float beta) {
    __shared__ half SA[BW * BW];
    __shared__ half SB[BW * BW];
    __shared__ float SC[BW * BW];

    float zero;
    half *ptr_zero = reinterpret_cast<half*>(&zero);
    ptr_zero[0] = __float2half(0.0f);
    ptr_zero[1] = __float2half(0.0f);
    float* SAfloat = reinterpret_cast<float *>(SA);
    float* SBfloat = reinterpret_cast<float *>(SB);
    const float* Afloat = reinterpret_cast<const float *>(A);
    const float* Bfloat = reinterpret_cast<const float *>(B);

    int tid = threadIdx.x;
    int warpId = threadIdx.x / 32;
    int warpRow = warpId / 4;
    int warpCol = warpId % 4;

    int numBlocksN = (N+BW-1)/BW;
    int numBlocksK = (K+BW-1)/BW;
    
    int blockId = blockIdx.x;
    int blockX = blockId / numBlocksN;
    int blockY = blockId % numBlocksN;

    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;
    wmma::fill_fragment(c_frag, 0.0f);

    for (int iter = 0; iter < numBlocksK; iter++) {
        // if k is even, able to load 2 halfs a time (20% speedup on nsight compute)
        if constexpr (isKEven) {
            // load block of A into shared memory
            for (int i = tid; i < BW * BW / 2; i += blockDim.x) {
                int row = i / (BW/2), col = i % (BW/2);
                row += blockX * BW;
                col += iter * (BW/2);
                int Aidx = row * K / 2 + col;
                SAfloat[i] = (Aidx >= M * K / 2) ? zero : Afloat[Aidx];
            }
            // load block of B into shared memory
            for (int i = tid; i < BW * BW / 2; i += blockDim.x) {
                int row = i / (BW/2), col = i % (BW/2);
                row += iter * BW;
                col += blockY * (BW/2);
                int Bidx = row * N / 2 + col;
                SBfloat[i] = (Bidx >= N * K / 2) ? zero : Bfloat[Bidx];
            }
        } else {
            // load block of A into shared memory
            for (int i = tid; i < BW * BW; i += blockDim.x) {
                int row = i / BW, col = i % BW;
                row += blockX * BW;
                col += iter * BW;
                SA[i] = (row >= M || col >= K) ? __float2half(0.0f) : A[row * K + col];
            }
            // load block of B into shared memory
            for (int i = tid; i < BW * BW; i += blockDim.x) {
                int row = i / BW, col = i % BW;
                row += iter * BW;
                col += blockY * BW;
                SB[i] = (row >= K || col >= N) ? __float2half(0.0f) : B[row * N + col];
            }
        }
        // make sure all loads finished
        __syncthreads();
        
        half *SA_warp_ptr = SA + 16 * BW * warpRow;
        half *SB_warp_ptr = SB + 16 * warpCol;
        for (int i = 0; i < 4; i++) {
            wmma::load_matrix_sync(a_frag, SA_warp_ptr, BW);
            wmma::load_matrix_sync(b_frag, SB_warp_ptr, BW);
            wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
            SA_warp_ptr += 16;
            SB_warp_ptr += 16 * BW;
        }
    }

    float *SC_warp_ptr = SC + 16 * BW * warpRow + 16 * warpCol;
    wmma::store_matrix_sync(SC_warp_ptr, c_frag, BW, wmma::mem_row_major);

    // finally store back to global
    __syncthreads();  // make sure all stores finished
    for (int i = tid; i < BW * BW; i += blockDim.x) {
        int row = i / BW;
        int col = i % BW;
        row += blockX * BW;
        col += blockY * BW;
        if (row < M && col < N) {
            C[row * N + col] = __float2half(beta * __half2float(C[row * N + col]) + alpha * SC[i]);
        }
    }
}

// A, B, and C are device pointers
void solve(const half* A, const half* B, half* C, int M, int N, int K, float alpha, float beta) {
    int numBlocksM = (M + BW -1) / BW;
    int numBlocksN = (N + BW - 1) / BW;
    if (K % 2 == 0) {
        matmul_kernel<true><<<numBlocksM * numBlocksN, 32 * NUM_WARPS>>>(A, B, C, M, N, K, alpha, beta);
    } else {
        matmul_kernel<false><<<numBlocksM * numBlocksN, 32 * NUM_WARPS>>>(A, B, C, M, N, K, alpha, beta);
    }
}