#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        float f = input[idx];
        output[idx] = f > 0 ? f : 0.01*f;
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    leaky_relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
}
